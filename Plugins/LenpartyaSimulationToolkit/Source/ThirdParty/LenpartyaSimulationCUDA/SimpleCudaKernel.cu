#if defined _WIN32 || defined _WIN64
#include <Windows.h>

#define LENPARTYACUDA_EXPORT __declspec(dllexport)
#else
#include <stdio.h>
#endif

#ifndef LENPARTYACUDA_EXPORT
#define LENPARTYACUDA_EXPORT
#endif


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__global__ void myKernel()
{
    printf("Hello from CUDA Kernel!\n");
}

LENPARTYACUDA_EXPORT void launchKernel()
{
    myKernel<<<1, 1>>>();
    hipDeviceSynchronize();

#if defined _WIN32 || defined _WIN64
    MessageBox(NULL, TEXT("Loaded LenpartyaCUDA.dll manually!"), TEXT("Third Party Plugin"), MB_OK);
#else
    printf("Loaded LenpartyaCUDA manually!");
#endif
}
