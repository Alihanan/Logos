#include "hip/hip_runtime.h"




#define LENPARTYACUDA_BUILD

#include "Public/LenpartyaSimulationCUDA/icosphere.h"
#include <hip/hip_runtime.h>




__global__ void kernel_compute_3d_coordinate(Coord3D* arrOut)
{


	return;
}

LENPARTYACUDA_EXPORT void compute_3d_coordinate(SimulationData* data)
{
	// grid and block dimensions
	dim3 blocks(1,1);
	dim3 threads(1,1);

	
	kernel_compute_3d_coordinate <<< blocks, threads >>> ();
	//return 0;
}

LENPARTYACUDA_EXPORT SimulationData* generate_simulation_struct(
	uint32_t NUM_DIVISIONS, double RADIUS
)
{
	SimulationData* ret;
	ret = new SimulationData(NUM_DIVISIONS, RADIUS);
	return ret;
}